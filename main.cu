#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "kernel.h"
#include "lodepng.h"

#include <iostream>
#include <chrono>
#include <vector>
#include <cassert>







/*
Class to calculate time taken by functions in seconds.
* Creating an object of the class in a function, calls the constructor which starts the timer.
* At the end of the function, the destructor is called which stops the timer and calculates the duration.
* We can get the duration manually using the getElapsedTime method.
*/
class Timer {
private:
	std::chrono::time_point<std::chrono::steady_clock> m_Start, m_End;
	std::chrono::duration<float> m_Duration;

public:
	Timer() {
		m_Start = std::chrono::high_resolution_clock::now();
	}

	~Timer() {
		m_End = std::chrono::high_resolution_clock::now();
		m_Duration = m_End - m_Start;

		std::cout << "Done (" << m_Duration.count() << " s)" << std::endl;
	}

	float getElapsedTime() {
		m_End = std::chrono::high_resolution_clock::now();
		m_Duration = m_End - m_Start;

		return m_Duration.count();
	}
};


// Display GPU info
// https://stackoverflow.com/a/5689133
void DisplayHeader() {
	const int kb = 1024;
	const int mb = kb * kb;
	std::cout << "NBody.GPU" << std::endl << "=========" << std::endl << std::endl;

	std::cout << "CUDA version:   v" << CUDART_VERSION << std::endl;

	int devCount;
	hipGetDeviceCount(&devCount);
	std::cout << "CUDA Devices: " << std::endl << std::endl;

	for (int i = 0; i < devCount; ++i) {
		hipDeviceProp_t props;
		hipGetDeviceProperties(&props, i);
		std::cout << i << ": " << props.name << ": " << props.major << "." << props.minor << std::endl;
		std::cout << "  Global memory:   " << props.totalGlobalMem / mb << "mb" << std::endl;
		std::cout << "  Shared memory:   " << props.sharedMemPerBlock / kb << "kb" << std::endl;
		std::cout << "  Constant memory: " << props.totalConstMem / kb << "kb" << std::endl;
		std::cout << "  Block registers: " << props.regsPerBlock << std::endl << std::endl;

		std::cout << "  Warp size:         " << props.warpSize << std::endl;
		std::cout << "  Threads per block: " << props.maxThreadsPerBlock << std::endl;
		std::cout << "  Max block dimensions: [ " << props.maxThreadsDim[0] << ", " << props.maxThreadsDim[1] << ", " << props.maxThreadsDim[2] << " ]" << std::endl;
		std::cout << "  Max grid dimensions:  [ " << props.maxGridSize[0] << ", " << props.maxGridSize[1] << ", " << props.maxGridSize[2] << " ]" << std::endl;
		std::cout << std::endl;
	}
}

std::vector<unsigned char> loadImage(const char* filename, unsigned& width, unsigned& height) {
	Timer timer;

	std::vector<unsigned char> pixels;

	unsigned error = lodepng::decode(pixels, width, height, filename);
	if (error) {
		std::cout << "Failed to load image: " << lodepng_error_text(error) << std::endl;
		std::cin.get();
		exit(-1);
	}

	return pixels;
}

std::vector<unsigned char> normalize(std::vector<unsigned> in, const unsigned width, const unsigned height) {
	std::vector<unsigned char> result(width * height * 4);

	unsigned char max = 0;
	unsigned char min = UCHAR_MAX;

	for (int i = 0; i < width * height; i++) {
		if (in[i] > max) {
			max = in[i];
		}

		if (in[i] < min) {
			min = in[i];
		}
	}

	// Normalize values to be between 0 and 255
	int mapIndex = 0;
	for (int i = 0; i < width * height * 4; i += 4, mapIndex++) {
		result[i] = result[i + 1] = result[i + 2] = (unsigned char)(255 * (in[mapIndex] - min) / (max - min));
		result[i + 3] = 255;
	}

	return result;
}


void CudaCall(const hipError_t& status) {
	if (status != hipSuccess) {
		std::cout << "Error [" << status << "]: " << hipGetErrorString(status) << " (" << __FILE__ << ":" << __LINE__ << ")" << std::endl;
	}
}

constexpr int scaleFactor = 2;

constexpr int minDisparity = 0;
constexpr int maxDisparity = 64;
constexpr int windowWidth = 11;
constexpr int windowHeight = 11;

constexpr int crossCheckingThreshold = 2;

constexpr int occlusionNeighbours = 256;

int main() {
	Timer timer;

	DisplayHeader();

	// Host variables
	std::vector<unsigned char> leftPixels, rightPixels;  // 1 byte: 0-255
	unsigned width, height, rightWidth, rightHeight;

	std::cout << "Reading Left Image...";
	leftPixels = loadImage("realL.png", width, height);

	std::cout << "Reading Right Image...";
	rightPixels = loadImage("realR.png", rightWidth, rightHeight);

	// left and right images are assumed to be of same dimensions
	assert(width == rightWidth && height == rightHeight);

	width /= scaleFactor;
	height /= scaleFactor;

	unsigned imSize = width * height;
	unsigned origSize = rightWidth * rightHeight;
	std::vector<unsigned> output(imSize);

	// Device variabels
	unsigned char* d_origL, * d_origR;
	unsigned* d_grayL, * d_grayR, * d_dispLR, * d_dispRL, * d_dispCC, * d_output;

	CudaCall(hipMalloc((void**)&d_origL, sizeof(unsigned char) * origSize * 4));  // 4: rgbd.
	CudaCall(hipMalloc((void**)&d_origR, sizeof(unsigned char) * origSize * 4));
	CudaCall(hipMalloc((void**)&d_grayL, sizeof(unsigned) * imSize));
	CudaCall(hipMalloc((void**)&d_grayR, sizeof(unsigned) * imSize));
	CudaCall(hipMalloc((void**)&d_dispLR, sizeof(unsigned) * imSize));
	CudaCall(hipMalloc((void**)&d_dispRL, sizeof(unsigned) * imSize));
	CudaCall(hipMalloc((void**)&d_dispCC, sizeof(unsigned) * imSize));
	CudaCall(hipMalloc((void**)&d_output, sizeof(unsigned) * imSize));

	// Copy Data from host to device
	CudaCall(hipMemcpy(d_origL, leftPixels.data(), sizeof(leftPixels[0]) * leftPixels.size(), hipMemcpyHostToDevice));
	CudaCall(hipMemcpy(d_origR, rightPixels.data(), sizeof(rightPixels[0]) * rightPixels.size(), hipMemcpyHostToDevice));

	// Profiling
	float elapsed = 0;
	hipEvent_t start, stop;

	CudaCall(hipEventCreate(&start));
	CudaCall(hipEventCreate(&stop));

	// Kernel Calls
	dim3 blocks(height / 21, width / 21);
	dim3 threads(21, 21);
	dim3 blocks1D((height / 21) * (width / 21));
	dim3 threads1D(21 * 21);

	// Scale and Gray left
	std::cout << "Converting Left Image to grayscale...";
	CudaCall(hipEventRecord(start));

	ScaleAndGray<<<blocks, threads>>>(d_origL, d_grayL, rightWidth, rightHeight, scaleFactor);

	CudaCall(hipEventRecord(stop));
	CudaCall(hipEventSynchronize(stop));
	CudaCall(hipEventElapsedTime(&elapsed, start, stop));
	std::cout << "Done (" << elapsed / 1000 << " s)" << std::endl;

	CudaCall(hipPeekAtLastError());
	CudaCall(hipDeviceSynchronize());

	// Scale and Gray right
	std::cout << "Converting Right Image to grayscale...";
	CudaCall(hipEventRecord(start));

	ScaleAndGray<<<blocks, threads>>>(d_origR, d_grayR, rightWidth, rightHeight, scaleFactor);

	CudaCall(hipEventRecord(stop));
	CudaCall(hipEventSynchronize(stop));
	CudaCall(hipEventElapsedTime(&elapsed, start, stop));
	std::cout << "Done (" << elapsed / 1000 << " s)" << std::endl;

	CudaCall(hipPeekAtLastError());
	CudaCall(hipDeviceSynchronize());

	// Disparity Left over Right
	std::cout << "Converting Left Disparity Map...";
	CudaCall(hipEventRecord(start));

	Zncc<<<blocks, threads>>>(d_grayL, d_grayR, d_dispLR, width, height, minDisparity, maxDisparity, windowWidth, windowHeight);

	CudaCall(hipEventRecord(stop));
	CudaCall(hipEventSynchronize(stop));
	CudaCall(hipEventElapsedTime(&elapsed, start, stop));
	std::cout << "Done (" << elapsed / 1000 << " s)" << std::endl;

	CudaCall(hipPeekAtLastError());
	CudaCall(hipDeviceSynchronize());

	// Disparity Right over Left
	std::cout << "Converting Right Disparity Map...";
	CudaCall(hipEventRecord(start));

	Zncc<<<blocks, threads>>>(d_grayR, d_grayL, d_dispRL, width, height, -maxDisparity, -minDisparity, windowWidth, windowHeight);

	CudaCall(hipEventRecord(stop));
	CudaCall(hipEventSynchronize(stop));
	CudaCall(hipEventElapsedTime(&elapsed, start, stop));
	std::cout << "Done (" << elapsed / 1000 << " s)" << std::endl;

	CudaCall(hipPeekAtLastError());
	CudaCall(hipDeviceSynchronize());

	// Cross Checking
	std::cout << "Performing Cross Checking...";
	CudaCall(hipEventRecord(start));

	CrossCheck<<<blocks1D, threads1D>>>(d_dispLR, d_dispRL, d_dispCC, imSize, crossCheckingThreshold);

	CudaCall(hipEventRecord(stop));
	CudaCall(hipEventSynchronize(stop));
	CudaCall(hipEventElapsedTime(&elapsed, start, stop));
	std::cout << "Done (" << elapsed / 1000 << " s)" << std::endl;

	CudaCall(hipPeekAtLastError());
	CudaCall(hipDeviceSynchronize());

	// Occlusion Filling
	std::cout << "Performing Occlusion Filling...";
	CudaCall(hipEventRecord(start));

	OcclusionFill<<<blocks, threads>>>(d_dispCC, d_output, width, height, occlusionNeighbours);

	CudaCall(hipEventRecord(stop));
	CudaCall(hipEventSynchronize(stop));
	CudaCall(hipEventElapsedTime(&elapsed, start, stop));
	std::cout << "Done (" << elapsed / 1000 << " s)" << std::endl;

	CudaCall(hipPeekAtLastError());
	CudaCall(hipDeviceSynchronize());

	// Copy data from device to host
	CudaCall(hipMemcpy(&output[0], d_output, sizeof(unsigned)* imSize, hipMemcpyDeviceToHost));

	lodepng::encode("output.png", normalize(output, width, height), width, height);

	std::cout << "The program took " << timer.getElapsedTime() << " s" << std::endl;

	hipFree(d_origL);
	hipFree(d_origR);
	hipFree(d_grayL);
	hipFree(d_grayR);
	hipFree(d_dispLR);
	hipFree(d_dispRL);
	hipFree(d_dispCC);
	hipFree(d_output);

	std::cin.get();
	return 0;
}


